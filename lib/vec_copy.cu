#include "hip/hip_runtime.h"
#include "vec_copy.hpp"

__global__ void copy_kernel(int  * const in, int *out, const unsigned int size){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if( i < size)
    out[i] = in[i];
}

int vec_copy(int * const in, int * out, unsigned int size){
  int * d_in, * d_out;

  hipError_t hipError_t;

  // assignment is wanted
  if( hipError_t = hipMalloc( (void **)&d_in, sizeof(int)*size) ){
      return hipError_t;
  }

  if( hipError_t = hipMalloc( (void **)&d_out, sizeof(int)*size) ){
    return hipError_t;
  }

  if( hipError_t = hipMemcpy(d_in, in, sizeof(int)*size, hipMemcpyHostToDevice ) ){
    return hipError_t;
  }

  copy_kernel<<<1,size>>>(d_in, d_out, size);
  if( hipError_t = hipGetLastError() ){
    return hipError_t;
  }

  if( hipError_t = hipMemcpy(out, d_out, sizeof(int)*size, hipMemcpyDeviceToHost ) ){
    return hipError_t;
  }

  hipFree(d_in);
  hipFree(d_out);
  
  return 0;
}
